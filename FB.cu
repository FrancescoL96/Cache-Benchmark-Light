#include "hip/hip_runtime.h"
#include <chrono>
#include <thread>
#include <iostream>
#include <random>
#include <cmath>
#include <atomic>

#include <stdio.h>

#include "Timer.cuh"
#include "CheckError.cuh"

#include <omp.h>

using namespace timer;

// Set PRINT to 1 for debug output
#define PRINT 1
#define FROM_debug 0
#define TO_debug 16

// Set ZERO to 1 to use Zero copy, set ZERO to 0 to use Unified Memory
#define ZERO 1

unsigned int N = 2;
const int POW = 16;
const float MINUTES = 0.1; // Dictates the length of the benchmark, but doesn't actually follow the length 
const int SUMS = 8;
const int BLOCK_SIZE_X = 512;
const int BLOCK_SIZE_Y = 1;


__global__
void gpu_compute(int* matrix, const int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    double fp0 = 2.0;
    double fp1 = 2.0;
	int mat = matrix[row];
	double res;
	for (int j = 0; j < 3; j++ ) {
		fp0 *= float(j) + atan(tgamma(sqrt(acosh(__ddiv_ru(3.14159265359 * mat, 0.7)))));
		fp1 += float(j) + tgamma(sqrt(acosh(__ddiv_ru(3.14159265359 * mat, 0.7))));
		fp0 *= float(j) * atan(tgamma(sqrt(acosh(__ddiv_ru(3.14159265359 * fp1, 0.7)))));
		fp1 *= float(j) / sqrt(tgamma(sqrt(acosh(__ddiv_ru(3.14159265359 * fp0, 0.7)))));
		res /= sqrt(fp0 + fp1);
	}
	if (17 % mat == 0) matrix[row] = res; // mat cannot be 17 or 1, so this statement always evaluates to false, forcing the compiler to actually execute the code in the for cycle (and not optimize it away)
}


void fill_data(int * d_matrix_host, int N){
	unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_int_distribution<int> distribution(1, 100);
    
    for (int i = 0; i < N; i++) {
		int temp = distribution(generator);
		if (temp == 17 || temp == 1) temp++;
		d_matrix_host[i] = temp;
    }
}

int main() {
    N = (unsigned int) pow(N, POW);
    int grid = N / BLOCK_SIZE_X;
    // -------------------------------------------------------------------------
    // DEVICE INIT
    dim3 DimGrid(grid, 1, 1);
    if (N % grid) DimGrid.x++;
    dim3 DimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);

    // -------------------------------------------------------------------------
    hipSetDeviceFlags(hipDeviceMapHost);
	
	Timer<HOST> TM;
	Timer<HOST> TM_update;
	Timer<HOST> TM_app;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    int * d_matrix_host;
    int * d_matrix;    

    #if ZERO
    // Zero Copy Allocation
	SAFE_CALL(hipHostAlloc((void **)&d_matrix_host, N * sizeof(int), hipHostMallocMapped));
    SAFE_CALL(hipHostGetDevicePointer((void **)&d_matrix, (void *) d_matrix_host , 0));
    #else
    // Unified Allocation    
  	SAFE_CALL(hipMallocManaged((void **)&d_matrix_host, N * sizeof(int)));
  	#endif    
    
    // -------------------------------------------------------------------------
    // MATRIX INITILIZATION
    std::cout << "Starting Initialization..." << std::endl;
	TM.start();
    fill_data(d_matrix_host, N);
    TM.stop();
    TM.print("Initialization Finished, time: ");

    // -------------------------------------------------------------------------
    // EXECUTION
    TM_app.start();
    std::cout << "Starting computation (GPU+CPU)..." << std::endl;
	for (int i = 0; i < int((MINUTES*60*1000)/33.3); i++) {
		TM.start();
	    gpu_compute << < DimGrid, DimBlock >> > (d_matrix_host, N);
	    #if !ZERO
		CHECK_CUDA_ERROR
		#endif
		TM_update.start();
		fill_data(d_matrix_host, N);
		TM_update.stop();
		
		#if ZERO
		CHECK_CUDA_ERROR
		#endif
		TM.stop();
    }
	#if ZERO
    CHECK_CUDA_ERROR
	#endif
	TM_app.stop();
	std::cout << "AVG UPDATE:  " << TM_update.total_duration()/int((MINUTES*60*1000)/33.3) << std::endl;
	if (ZERO) 
		TM_app.print("App run time ZC: ");
	else 
		TM_app.print("App run time UM: ");
	std::cout << "AVG APP:  " << TM_app.duration()/int((MINUTES*60*1000)/33.3) << std::endl;
    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    #if ZERO
    SAFE_CALL(hipHostFree(d_matrix));
    #else
    SAFE_CALL(hipFree(d_matrix_host));
    #endif
    
    // -------------------------------------------------------------------------
    hipDeviceReset();

}

